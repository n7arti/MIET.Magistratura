#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>

#define N 1024 // ������ �������
#define BLOCK_SIZE (N/2) 

__device__ void Comparator(unsigned int& keyA, unsigned int& valA, unsigned int& keyB, unsigned int& valB, unsigned int dir)
{
    unsigned int t;
    if ((valA > valB) == dir) //�������� ������� (keyA, valA) � (keyB, valB)
    {
        t = keyA; 
        keyA = keyB; 
        keyB = t;
        t = valA; 
        valA = valB; 
        valB = t;
    }
}
__global__ void bitonicSortShared(unsigned int* dstKey, unsigned int* dstVal, unsigned int* srcKey, unsigned int* srcVal, unsigned int arrayLength, unsigned int dir)
{
    __shared__ unsigned int sk[BLOCK_SIZE * 2];
    __shared__ unsigned int sv[BLOCK_SIZE * 2];
    int index = blockIdx.x * BLOCK_SIZE * 2 + threadIdx.x;

    sk[threadIdx.x] = srcKey[index]; sv[threadIdx.x] = srcVal[index];
    sk[threadIdx.x + BLOCK_SIZE] = srcKey[index + BLOCK_SIZE];  sv[threadIdx.x + BLOCK_SIZE] = srcVal[index + BLOCK_SIZE];

    for (unsigned int size = 2; size < arrayLength; size <<= 1)
    {//������������ �������
        unsigned int ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);
        for (unsigned int stride = size >> 1; stride > 0; stride >>= 1)
        {
            __syncthreads();
            unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(sk[pos], sv[pos], sk[pos + stride], sv[pos + stride], ddd);
        }
    }
    //��������� ��� - ������������ �������
    for (unsigned int stride = arrayLength >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        Comparator(sk[pos], sv[pos], sk[pos + stride], sv[pos + stride], dir);
    }
    __syncthreads();

    dstKey[index] = sk[threadIdx.x]; dstVal[index] = sv[threadIdx.x];
    dstKey[index + BLOCK_SIZE] = sk[threadIdx.x + BLOCK_SIZE]; dstVal[index + BLOCK_SIZE] = sv[threadIdx.x + BLOCK_SIZE];
}

int main() {
    int* a = (int*)malloc(N * sizeof(int));
    int* result_GPU = (int*)malloc(N * sizeof(int));
    int* result_CPU = (int*)malloc(N * sizeof(int));

    // ��������� ������ �� GPU
    int* dev_a, * dev_result_GPU;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_result_GPU, N * sizeof(int));

    hipEvent_t start, stop;		//��������� ���������� ����  hipEvent_t 
    float       gpuTime = 0.0f;
    // ������� ������� ������ � ��������� ���������� ���� 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ������������� ������� ���������� ����������
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 100;
    }

    // ���������� ����� �� CPU
    for (int i = 0; i < N-1; i++) {
        Comparator(i, a[i], i + 1, a[i + 1], 1);
    }

    // ����������� ������ � ����� �� ����������
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

    //����������� ������� start  � ������� ����� 
    hipEventRecord(start, 0);

    // ������ ���� CUDA ��� ���������� ���������� �����
    bitonicSortShared << <(N + 255) / 256, 256 >> > (dev_a, dev_result_GPU);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // ����������� ����� ����� ��������� 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.5f ms\n", gpuTime);

    // ����������� ����������� � ���������� �� ����
    hipMemcpy(result_GPU, dev_result_GPU, N * sizeof(int), hipMemcpyDeviceToHost);

    // �������� ����������� ���������� �����
    for (int i = 0; i < N; i++) {
        if (result_CPU[i] != result_GPU[i]) {
            printf("Verification failed at element %d!\n", i);
        }
    }

    // ������������ ������
    free(a);
    free(result_GPU);
    free(result_CPU);
    hipFree(dev_a);
    hipFree(dev_result_GPU);

    // ���������� ��������� ������� 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}