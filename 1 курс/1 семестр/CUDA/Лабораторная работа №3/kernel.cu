#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>

#define N 1024 // Размер массива
#define BLOCK_SIZE 16
#define LOG_NUM_BANKS 4 
#define CONLICT_FREE_OFFS(i) ((i) >> LOG_NUM_BANKS) 

// CUDA Kernel для префиксной суммы
__global__ void scan(float* a, float* result, float* sums, int n) {
    __shared__ float temp[2 * BLOCK_SIZE + CONFLICT_FREE_OFFS(2 * BLOCK_SIZE)];
    int tid = threadIdx.x;
    int offset = 1;
    int ai = tid;
    int bi = tid + (n / 2);
    int offsA = CONFLICT_FREE_OFFS(ai);
    int offsB = CONFLICT_FREE_OFFS(bi);
    temp[ai + offsA] = a[ai + 2 * BLOCK_SIZE * blockIdx.x];
    temp[bi + offsB] = a[bi + 2 * BLOCK_SIZE * blockIdx.x];
    for (int d = n >> 1; d > 0; d >>= 1, offset <<= 1)
    {
        __syncthreads();
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFS(ai);
            bi += CONFLICT_FREE_OFFS(bi);
            temp[bi] += temp[ai];
        }
    }
    if (tid == 0)
    {
        int i = n - 1 + CONFLICT_FREE_OFFS(n-1);  // для scan больших массивов
        sums [blockIdx.x] = temp [i];             // для scan больших массивов
        temp[i] = 0; // clear the last element 
    }
    for (int d = 1; d < n; d <<= 1)
    {
        offset >>= 1;
        __syncthreads();
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            float t;
            ai += CONFLICT_FREE_OFFS(ai);
            bi += CONFLICT_FREE_OFFS(bi);
            t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    result[ai + 2 * BLOCK_SIZE * blockIdx.x] = temp[ai + offsA];
    result[bi + 2 * BLOCK_SIZE * blockIdx.x] = temp[bi + offsB];
}

__global__ void scanDistribute(float* data, float* sums)
{
    data[threadIdx.x + blockIdx.x * 2 * BLOCK_SIZE] += sums[blockIdx.x];
}
void scanOnGPU(float* a, float* result, int n)
{
    int numBlocks = n / (2 * BLOCK_SIZE);
    float* sums; // суммы элементов для каждого блока 
    float* sums2; // результаты scan этих сумм 
    if (numBlocks < 1) numBlocks = 1;
    // выделяем память под массивы 
    hipMalloc((void**)&sums, numBlocks * sizeof(float));
    hipMalloc((void**)&sums2, numBlocks * sizeof(float));
    // поблочный scan
    dim3 threads(BLOCK_SIZE, 1, 1), blocks(numBlocks, 1, 1); 	scan << <blocks, threads >> > (inData, outData, sums, 2 * BLOCK_SIZE);
    // выполняем scan для сумм 
    if (n >= 2 * BLOCK_SIZE)
        scanOnCPU(sums, sums2, numBlocks);
    else hipMemcpy(sums2, sums, numBlocks * sizeof(float), hipMemcpyDeviceToDevice);
    // корректируем результат 
    threads = dim3(2 * BLOCK_SIZE, 1, 1);
    blocks = dim3(numBlocks - 1, 1, 1);
    scanDistribute << <blocks, threads >> > (outData + 2 * BLOCK_SIZE, sums2 + 1);
    hipFree(sums);
    hipFree(sums2);
}


void scanOnCPU(float* a, float* result, int n) {
    result[0] = 0;
    for (int i = 1; i < n; i++)
        result[i] = result[i - 1] + a[i - 1];

}

int main() {
    float* a = (float*)malloc(N * sizeof(float));
    float* result_GPU = (float*)malloc(N * sizeof(float));
    float* result_CPU = (float*)malloc(N * sizeof(float));

    // Выделение памяти на GPU
    float* dev_a, * dev_result_GPU;
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_result_GPU, N * sizeof(float));

    hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
    float       gpuTime = 0.0f;
    // создаем события начала и окончания выполнения ядра 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Инициализация массива случайными значениями
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 100;
    }

    // Префиксная сумма на CPU
    scanOnCPU(a, result_CPU, N);

    // Копирование данных с хоста на устройство
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    //привязываем событие start  к данному месту 
    hipEventRecord(start, 0);

    // Запуск ядра CUDA для вычисления префиксной суммы
    scanOnGPU << <(N + 255) / 256, 256 >> > (dev_a, dev_result_GPU);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // запрашиваем время между событиями 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.5f ms\n", gpuTime);

    // Копирование результатов с устройства на хост
    hipMemcpy(result_GPU, dev_result_GPU, N * sizeof(float), hipMemcpyDeviceToHost);

    // Проверка результатов префиксной суммы
    for (int i = 0; i < N; i++) {
        if (result_CPU[i] != result_GPU[i]) {
            printf("Verification failed at element %d!\n", i);
        }
    }

    // Освобождение памяти
    free(a);
    free(result_GPU);
    free(result_CPU);
    hipFree(dev_a);
    hipFree(dev_result_GPU);

    // уничтожаем созданные события 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}