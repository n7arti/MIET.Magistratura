#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#define N 1024

// ������� �� CPU ��� ����������� ����������
void multiplyOnCPU(float* a, float* b, float* c) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] * b[i];
    }
}

__global__ void multiplyOnGPU(float* a, float* b, float* c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() { 
    float* a = (float*)malloc(N * sizeof(float));
    float* b = (float*)malloc(N * sizeof(float));
    float* � = (float*)malloc(N * sizeof(float));

    float* dev_a, * dev_b, * dev_c;

    // �������� ������ �� GPU
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));

    hipEvent_t start, stop;		//��������� ���������� ����  hipEvent_t 
    float       gpuTime = 0.0f;
    // ������� ������� ������ � ��������� ���������� ���� 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ������������� �������� �� �����
    for (int i = 0; i < N; i++) {
        a[i] = i; // �������� ������� ���������� ����������
        b[i] = i+1;
    }

    // ����������� ������ � ����� �� ����������
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    //����������� ������� start  � ������� ����� 
    hipEventRecord(start, 0);
    // ������ ���� �� GPU
    multiplyOnGPU << <dim3((N / 512), 1), dim3(512, 1) >> > (d_a, d_b, d_c);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // ����������� ����� ����� ��������� 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
   

    // ����������� ���������� � ���������� �� ����
    hipMemcpy(�, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // ����������� ���������� �� CPU
    multiplyOnCPU(a, b, �);

    // �������� �����������
    for (int i = 0; i < N; i++) {
        if (�[i] != a[i] * b[i]) {
            printf("������: c[%d] = %.2f, ��������� %.2f\n", i, �[i], a[i] * b[i]);
            break;
        }
    }

    // ������������ ������
    free(a);
    free(b);
    free(�);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // ���������� ��������� ������� 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}