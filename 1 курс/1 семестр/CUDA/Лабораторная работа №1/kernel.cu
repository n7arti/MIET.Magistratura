#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#define	N	(1024*1024)

__global__ void kernel(float* data)
{
    int   idx = blockIdx.x * blockDim.x + threadIdx.x;
    float x = 2.0f * 3.1415926f * (float)idx / (float)N;
    data[idx] = sinf(sqrtf(x));
}

void deviceQuery() 
{
    int		deviceCount;
    hipDeviceProp_t	devProp;

    hipGetDeviceCount(&deviceCount);
    printf("Found %d devices\n", deviceCount);

    for (int device = 0; device < deviceCount; device++)
    {
        hipGetDeviceProperties(&devProp, device);
        printf("Device %d\n", device);
        printf("Compute capability     : %d.%d\n", devProp.major, devProp.minor);
        printf("Name                   : %s\n", devProp.name);
        printf("Total Global Memory    : %u\n", devProp.totalGlobalMem);
        printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
        printf("Registers per block    : %d\n", devProp.regsPerBlock);
        printf("Warp size              : %d\n", devProp.warpSize);
        printf("Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
        printf("Total constant memory  : %d\n", devProp.totalConstMem);
    }

}

int main(int argc, char* argv[])
{

    deviceQuery(); // Получение информации об устройстве и вывод в консоль

    float* a = (float*)malloc(N * sizeof(float));
    float* dev = nullptr;
    // выделить память на GPU
    hipMalloc((void**)&dev, N * sizeof(float));

    hipDeviceProp_t deviceProp; // Определение структуры hipDeviceProp_t
    hipGetDeviceProperties(&deviceProp, 0);

    hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
    float       gpuTime = 0.0f;
    // создаем события начала и окончания выполнения ядра 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Определение максимального количества блоков на мультипроцессор и максимального размера сетки
    int maxBlocksPerGrid;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocksPerGrid, kernel, 512, 0);
    int maxGridSize = maxBlocksPerGrid * deviceProp.multiProcessorCount;

    dim3 gridSize(maxGridSize, 1);
    dim3 blockSize(512, 1);

    //привязываем событие start  к данному месту 
    hipEventRecord(start, 0);

    // конфигурация запуска N нитей
    kernel << <gridSize, blockSize >> > (dev);

    // скопировать результаты в память CPU
    hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);

    // освободить выделенную память
    hipFree(dev);
    free(a);
    for (int idx = 0; idx < N; idx++)
        printf("a[%d] = %.5f\n", idx, a[idx]);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // запрашиваем время между событиями 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
    // уничтожаем созданные события 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
