#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>

#define N 1024
#define BLOCK_SIZE 16 
void multiplyOnCPU(float* a, float* b, float* c) {
    for (int i = 0; i < N; i++) {
        if (c[i] != a[i] * b[i]) {
            printf("Verification failed at element %d!\n", i);
        }
    }
}

__global__ void multiplyOnGPU(float* a, float* b, float* c, int n) {
    int bx = blockIdx.x;  // индексы блока 
    int by = blockIdx.y;  // 

    int tx = threadIdx.x;  // индексы нити внутри блока 
    int ty = threadIdx.y;  // 

    float sum = 0.0f;

    // смещение для a[i][0]
    int ia = n * BLOCK_SIZE * by + n * ty;

    // смещение для b[0][i]
    int ib = BLOCK_SIZE * bx + tx;

    // смещение для результата 
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

    // перемножаем и суммируем 
    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k * n];
    c[ic + n * ty + tx] = sum; // запоминаем результат 

}

int main() {
    float* a = (float*)malloc(N * sizeof(float));
    float* b = (float*)malloc(N * sizeof(float));
    float* с = (float*)malloc(N * sizeof(float));

    float* dev_a, * dev_b, * dev_c;

    // выделить память на GPU
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));

    hipEvent_t start, stop;		//описываем переменные типа  hipEvent_t 
    float       gpuTime = 0.0f;
    // создаем события начала и окончания выполнения ядра 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Инициализация векторов на хосте
    for (int i = 0; i < N; i++) {
        a[i] = i; // Заполним векторы случайными значениями
        b[i] = i + 1;
    }

    // Копирование данных с хоста на устройство
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Запуск ядра CUDA с блоками и потоками
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    //привязываем событие start  к данному месту 
    hipEventRecord(start, 0);
    // Запуск ядра на GPU
    multiplyOnGPU << <gridSize, blockSize >> > (dev_a, dev_b, dev_c, N);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // запрашиваем время между событиями 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.5f ms\n", gpuTime);


    // Копирование результата с устройства на хост
    hipMemcpy(с, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Верификация результата на CPU
    multiplyOnCPU(a, b, с);

    // Освобождение памяти
    free(a);
    free(b);
    free(с);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // уничтожаем созданные события 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}