#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>

#define N 1024
#define BLOCK_SIZE 16 
void multiplyOnCPU(float* a, float* b, float* c) {
    for (int i = 0; i < N*N; i++) {
        if (c[i] != a[i] * b[i]) {
            printf("Verification failed at element %d!\n", i);
        }
    }
}

__global__ void multiplyOnGPUShared(float* a, float* b, float* c, int n) {
    int bx = blockIdx.x;  // ������� ����� 
    int by = blockIdx.y;  // 

    int tx = threadIdx.x;  // ������� ���� ������ ����� 
    int ty = threadIdx.y;  // 

    int	aBegin = n * BLOCK_SIZE * by;
    int	aEnd = aBegin + n - 1;
    int aStep = BLOCK_SIZE;
    int	bBegin = bx * BLOCK_SIZE;
    int bStep = BLOCK_SIZE * n;
    float	sum = 0.0f;
    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
    {
        __shared__ float	as[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float	bs[BLOCK_SIZE][BLOCK_SIZE];
        as[ty][tx] = a[ia + n * ty + tx];
        bs[ty][tx] = b[ib + n * ty + tx];
        __syncthreads(); // ��������, ��� ���������� ��������� ��������� 
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += as[ty][k] * bs[k][tx];
        __syncthreads(); // ��������, ��� ���������� ������ ������ �� ����� 
    }
    c[n * BLOCK_SIZE * by + BLOCK_SIZE * bx + n * ty + tx] = sum;

}

__global__ void multiplyOnGPUGlobal(float* a, float* b, float* c, int n) {
    int bx = blockIdx.x;  // ������� ����� 
    int by = blockIdx.y;  // 

    int tx = threadIdx.x;  // ������� ���� ������ ����� 
    int ty = threadIdx.y;  // 

    float sum = 0.0f;

    // �������� ��� a[i][0]
    int ia = n * BLOCK_SIZE * by + n * ty;

    // �������� ��� b[0][i]
    int ib = BLOCK_SIZE * bx + tx;

    // �������� ��� ���������� 
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

    // ����������� � ��������� 
    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k * n];
    c[ic + n * ty + tx] = sum; // ���������� ��������� 

}

int main() {
    float* a = (float*)malloc(N * sizeof(float));
    float* b = (float*)malloc(N * sizeof(float));
    float* � = (float*)malloc(N * sizeof(float));

    float* dev_a, * dev_b, * dev_c;
    size_t size = N * N * sizeof(float);

    // �������� ������ �� GPU
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b,size);
    hipMalloc((void**)&dev_c, size);

    hipEvent_t start, stop;		//��������� ���������� ����  hipEvent_t 
    float       gpuTime = 0.0f;
    // ������� ������� ������ � ��������� ���������� ���� 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ������������� �������� �� �����
    for (int i = 0; i < N*N; i++) {
        a[i] = i; // �������� ������� ���������� ����������
        b[i] = i + 1;
    }

    // ����������� ������ � ����� �� ����������
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // ������ ���� CUDA � ������� � ��������
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    //����������� ������� start  � ������� ����� 
    hipEventRecord(start, 0);
    // ������ ���� �� GPU
    multiplyOnGPUGlobal << <blocksPerGrid, threadsPerBloc >> > (dev_a, dev_b, dev_c, N*N);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // ����������� ����� ����� ��������� 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU Global: %.5f ms\n", gpuTime);

    hipEventRecord(start, 0);
    // ������ ���� �� GPU
    multiplyOnGPUShared << <blocksPerGrid, threadsPerBloc >> > (dev_a, dev_b, dev_c, N * N);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    // ����������� ����� ����� ��������� 
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU Shared: %.5f ms\n", gpuTime);


    // ����������� ���������� � ���������� �� ����
    hipMemcpy(�, dev_c, size, hipMemcpyDeviceToHost);

    // ����������� ���������� �� CPU
    multiplyOnCPU(a, b, �);

    // ������������ ������
    free(a);
    free(b);
    free(�);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // ���������� ��������� ������� 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}