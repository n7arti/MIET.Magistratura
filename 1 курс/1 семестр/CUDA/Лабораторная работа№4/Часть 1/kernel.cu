#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "../Common/hip/hip_vector_types.h"
#include "../Common/helper_functions.h"
#define STB_IMAGE_IMPLEMENTATION   
#include "../stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../stb/stb_image_write.h"
#include <stdio.h>

__global__ void transformKernel(float* output,
    hipTextureObject_t texObj,
    int width, int height,
    float theta)
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    // Transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    // Read from texture and write to global memory
    output[y * width + x] = tex2D<float>(texObj, tu, tv);
}

char* toBinary(int n, int len)
{
    char* binary = (char*)malloc(sizeof(char) * len);
    int k = 0;
    for (unsigned i = (1 << len - 1); i > 0; i = i / 2) {
        binary[k++] = (n & i) ? '1' : '0';
    }
    binary[k] = '\0';
    return binary;
}

void print_binary(unsigned char* n)
{
    int len = 8;
    char* binary = toBinary(n[0], len);
    printf("The binary representation of %d is %s\n", n[0], binary);
}

void print_binary(float* n)
{
    int len = 32;
    char* binary = toBinary(n[0], len);
    printf("The binary representation of %d is %s\n", n[0], binary);
}

int main()
{
    int height = 320;
    int width = 213;
    int texChannels;
    float angle = 45;

    //Загружаем выбранную картинку
    stbi_uc* pixels = stbi_load("cat.bmp", &width, &height, &texChannels, STBI_grey);

    // Allocate and set some host data
    float* h_data = (float*)malloc(sizeof(float) * width * height);
    for (int i = 0; i < height * width; ++i)
        h_data[i] = (pixels[i] & 0xff);

    print_binary(pixels);
    print_binary(h_data);

    //Выделение памяти и копирование информации на видеокарту
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // Allocate result of transformation in device memory
    float* output;
    hipMalloc(&output, width * height * sizeof(float));

    // Set pitch of the source (the width in memory in bytes of the 2D array pointed
    // to by src, including padding), we dont have any padding
    const size_t spitch = width * sizeof(float);
    // Copy data located at address h_data in host memory to device memory
    hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float),
        height, hipMemcpyHostToDevice);

    //Настройка переданной текстуры
    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    
    // Вызов функции переворота
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
        (height + threadsperBlock.y - 1) / threadsperBlock.y);
    transformKernel <<< numBlocks, threadsperBlock >>>(output, texObj, width, height, angle);
    
    // Возврат данных обратно в оперативку
    hipMemcpy(h_data, output, width * height * sizeof(float),
        hipMemcpyDeviceToHost);

    unsigned char* h_data_char = (unsigned char*)malloc(sizeof(unsigned char) * width * height);
  
    print_binary(h_data);

    for (int i = 0; i < height * width; ++i)
    {
        h_data_char[i] = (unsigned char)h_data[i];
    }
        
    print_binary(h_data_char);

    stbi_write_jpg("cat-out.jpg", width, height, 1, h_data_char, 100);
    
    // Destroy texture object
    hipDestroyTextureObject(texObj);

    // Free device memory
    hipFreeArray(cuArray);
    hipFree(output);
    stbi_image_free(pixels);
    // Free host memory
    free(h_data);

    return 0;
}